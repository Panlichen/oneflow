
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

#ifdef WITH_CUTLASS

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/ep/cuda/cuda_stream.h"
#include "oneflow/core/ep/include/primitive/permute.h"
#include "cutlass/gemm/warp/mma.h"
#include "kernel_forward.h"
#include "oneflow/core/kernel/cuda_graph_support.h"
#include "oneflow/user/kernels/fmha_flash_attention/fmha.h"
#include "oneflow/user/kernels/fmha_flash_attention/include/fmha_flash_attention.h"

namespace oneflow {

namespace user_op {

namespace {

template<typename T, int pack_size>
struct alignas(pack_size * sizeof(T)) Pack {
  T elem[pack_size];
};

template<typename T>
__global__ void PackQkv(int b, int s, int nh, int d, const T* q, const T* k, const T* v, T* o,
                        int32_t* seq_len) {
  int count = b * s * nh * d * 3;
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < count; i += blockDim.x * gridDim.x) {
    int row = i / (d * 3);
    int out_col = i - row * (d * 3);
    T out;
    if (out_col < d) {
      out = q[row * d + out_col];
    } else if (out_col < 2 * d) {
      out = k[row * d + out_col - d];
    } else {
      out = v[row * d + out_col - d * 2];
    }
    o[i] = out;
  }
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < b + 1; i += blockDim.x * gridDim.x) {
    seq_len[i] = i * s;
  }
}

struct Params {
  DataType data_type;
  int64_t num_batches;
  int64_t num_heads;
  int64_t query_seq_len;
  int64_t kv_seq_len;
  int64_t head_size;
  int64_t value_head_size;
  int64_t query_hidden_stride;
  int64_t key_hidden_stride;
  int64_t value_hidden_stride;
  bool causal;
  const void* query_ptr;
  const void* key_ptr;
  const void* value_ptr;
  void* out_ptr;
  void* workspace;
  int64_t workspace_size;
};

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block,
         bool single_value_iteration>
void LaunchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  using Attention = AttentionKernel<T, ArchTag, is_aligned, queries_per_block, keys_per_block,
                                    single_value_iteration>;
  typename Attention::Params p;
  p.query_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.query_ptr));
  p.key_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.key_ptr));
  p.value_ptr = const_cast<T*>(reinterpret_cast<const T*>(params.value_ptr));
  p.logsumexp_ptr = nullptr;
  p.output_ptr = reinterpret_cast<T*>(params.out_ptr);
  if (Attention::kNeedsOutputAccumulatorBuffer) {
    using Acc = typename Attention::accum_t;
    CHECK_GE(params.workspace_size, params.num_batches * params.query_seq_len * params.num_heads
                                        * params.value_head_size * sizeof(Acc));
    p.output_accum_ptr = reinterpret_cast<Acc*>(params.workspace);
  } else {
    p.output_accum_ptr = nullptr;
  }
  p.num_heads = params.num_heads;
  p.num_batches = params.num_batches;
  p.head_dim = params.head_size;
  p.head_dim_value = params.value_head_size;
  p.num_queries = params.query_seq_len;
  p.num_keys = params.kv_seq_len;
  p.q_strideM = params.query_hidden_stride;
  p.k_strideM = params.key_hidden_stride;
  p.v_strideM = params.value_hidden_stride;
  p.o_strideM = p.num_heads * params.value_head_size;

  p.q_strideH = params.head_size;
  p.k_strideH = params.head_size;
  p.v_strideH = params.value_head_size;
  p.o_strideH = params.value_head_size;

  p.q_strideB = params.query_seq_len * p.q_strideM;
  p.k_strideB = params.kv_seq_len * p.k_strideM;
  p.v_strideB = params.kv_seq_len * p.v_strideM;
  p.o_strideB = params.query_seq_len * p.o_strideM;

  p.causal = params.causal;

  constexpr auto kernel_fn = attention_kernel_batched_impl<Attention>;
  int smem_bytes = sizeof(typename Attention::SharedStorage);
  if (smem_bytes > 0xc000) {
    static bool once = [&]() {
      cudaFuncSetAttribute(kernel_fn, cudaFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
      return true;
    }();
  }
  CHECK(Attention::check_supported(p));
  kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes, stream->cuda_stream()>>>(p);
}

template<typename T, typename ArchTag, bool is_aligned, int queries_per_block, int keys_per_block>
void DispatchSingleValueIteration(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= keys_per_block) {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block, true>(params,
                                                                                       stream);
  } else {
    LaunchCutlassFmha<T, ArchTag, is_aligned, queries_per_block, keys_per_block, false>(params,
                                                                                        stream);
  }
}

template<typename T, typename ArchTag, bool is_aligned>
void DispatchKeysPerBlock(const Params& params, ep::CudaStream* stream) {
  if (params.value_head_size <= 64) {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 64, 64>(params, stream);
  } else {
    DispatchSingleValueIteration<T, ArchTag, is_aligned, 32, 128>(params, stream);
  }
}

template<typename T, typename ArchTag>
void DispatchIsAligned(const Params& params, ep::CudaStream* stream) {
  if (reinterpret_cast<uintptr_t>(params.query_ptr) % 16 == 0
      && reinterpret_cast<uintptr_t>(params.key_ptr) % 16 == 0
      && params.query_hidden_stride % (16 / sizeof(T)) == 0
      && params.key_hidden_stride % (16 / sizeof(T)) == 0) {
    DispatchKeysPerBlock<T, ArchTag, true>(params, stream);
  } else {
    DispatchKeysPerBlock<T, ArchTag, false>(params, stream);
  }
}

template<typename T>
void DispatchArchTag(const Params& params, ep::CudaStream* stream) {
  const int major = stream->device_properties().major;
  const int minor = stream->device_properties().minor;

  if (major == 8) {
    DispatchIsAligned<T, cutlass::arch::Sm80>(params, stream);
  } else if (major == 7) {
    if (minor == 5) {
      DispatchIsAligned<T, cutlass::arch::Sm75>(params, stream);
    } else {
      DispatchIsAligned<T, cutlass::arch::Sm70>(params, stream);
    }
  } else {
    UNIMPLEMENTED();
  }
}

void DispatchCutlassFmha(const Params& params, ep::CudaStream* stream) {
  if (params.data_type == DataType::kFloat16) {
    DispatchArchTag<cutlass::half_t>(params, stream);
  } else if (params.data_type == DataType::kFloat) {
    DispatchArchTag<cutlass::tfloat32_t>(params, stream);
  } else {
    UNIMPLEMENTED();
  }
}

class FusedMultiHeadAttentionInferenceKernel final : public user_op::OpKernel,
                                                     public user_op::CudaGraphSupport {
 public:
  FusedMultiHeadAttentionInferenceKernel() = default;
  ~FusedMultiHeadAttentionInferenceKernel() override = default;

 private:
  using user_op::OpKernel::Compute;
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const Tensor* query = ctx->Tensor4ArgNameAndIndex("query", 0);
    const Tensor* key = ctx->Tensor4ArgNameAndIndex("key", 0);
    const Tensor* value = ctx->Tensor4ArgNameAndIndex("value", 0);
    Tensor* out = ctx->Tensor4ArgNameAndIndex("out", 0);
    Tensor* tmp = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);
    const DataType data_type = query->data_type();
    CHECK_EQ(key->data_type(), data_type);
    CHECK_EQ(value->data_type(), data_type);
    CHECK_EQ(out->data_type(), data_type);
    CHECK_EQ(query->shape_view().NumAxes(), 3);
    CHECK_EQ(key->shape_view().NumAxes(), 3);
    CHECK_EQ(value->shape_view().NumAxes(), 3);
    CHECK_EQ(out->shape_view().NumAxes(), 3);
    const int64_t batch_size = query->shape_view().At(0);
    CHECK_EQ(key->shape_view().At(0), batch_size);
    CHECK_EQ(value->shape_view().At(0), batch_size);
    CHECK_EQ(out->shape_view().At(0), batch_size);
    const int64_t query_seq_len = query->shape_view().At(1);
    CHECK_EQ(out->shape_view().At(1), query_seq_len);
    const int64_t kv_seq_len = key->shape_view().At(1);
    CHECK_EQ(value->shape_view().At(1), kv_seq_len);
    const int64_t num_heads = ctx->Attr<int64_t>("num_heads");
    const bool causal = ctx->Attr<bool>("causal");

    const auto ParseHiddenDim = [&](const std::string& tag, const ShapeView& shape,
                                    int64_t* hidden_slice_start, int64_t* hidden_size) {
      *hidden_slice_start = ctx->Attr<int64_t>(tag + "_hidden_slice_start");
      CHECK_GE(*hidden_slice_start, 0);
      int64_t hidden_slice_end = ctx->Attr<int64_t>(tag + "_hidden_slice_end");
      if (hidden_slice_end < 0) { hidden_slice_end = hidden_slice_end + shape.At(2) + 1; }
      CHECK_GT(hidden_slice_end, 0);
      CHECK_LE(hidden_slice_end, shape.At(2));
      CHECK_GT(hidden_slice_end, *hidden_slice_start);
      *hidden_size = hidden_slice_end - *hidden_slice_start;
      CHECK_EQ(*hidden_size % num_heads, 0);
    };

    int64_t query_hidden_offset = 0;
    int64_t query_hidden_size = 0;
    ParseHiddenDim("query", query->shape_view(), &query_hidden_offset, &query_hidden_size);

    int64_t key_hidden_offset = 0;
    int64_t key_hidden_size = 0;
    ParseHiddenDim("key", key->shape_view(), &key_hidden_offset, &key_hidden_size);
    CHECK_EQ(key_hidden_size, query_hidden_size);

    int64_t value_hidden_offset = 0;
    int64_t value_hidden_size = 0;
    ParseHiddenDim("value", value->shape_view(), &value_hidden_offset, &value_hidden_size);

    CHECK_EQ(out->shape_view().At(2), value_hidden_size);

    auto* cuda_stream = ctx->stream()->As<ep::CudaStream>();

    const static bool enable_trt_flash_attn =
        ParseBooleanFromEnv("ONEFLOW_KERENL_FMHA_ENABLE_TRT_FLASH_ATTN_IMPL", false)
        && ParseBooleanFromEnv("ONEFLOW_MATMUL_ALLOW_HALF_PRECISION_ACCUMULATION", false);
    const int arch = cuda_stream->cuda_arch() / 10;
    const bool inputs_contiguous =
        query_hidden_offset == 0 && query_hidden_size == query->shape_view().At(2)
        && key_hidden_offset == 0 && key_hidden_size == key->shape_view().At(2)
        && value_hidden_offset == 0 && value_hidden_size == value->shape_view().At(2);
    const bool is_trt_supported_arch = (arch == 80 || arch == 86 || arch == 89);
    const int64_t query_head_size = query_hidden_size / num_heads;
    const bool is_trt_supported_head_size = ((query_head_size == 40) || (query_head_size == 64));
    // Avoid PackQKV overhead when seq_len is small.
    const bool is_long_seq_len = query_seq_len >= 512;
    if (enable_trt_flash_attn && inputs_contiguous && data_type == DataType::kFloat16
        && query_seq_len == kv_seq_len && query_hidden_size == value_hidden_size
        && is_trt_supported_head_size && is_long_seq_len && is_trt_supported_arch && (!causal)) {
      // The fmha implementation below is based on TensorRT's multiHeadFlashAttentionPlugin
      // implementation at:
      // https://github.com/NVIDIA/TensorRT/tree/main/plugin/multiHeadFlashAttentionPlugin
      int32_t cu_seqlens_d_size = (batch_size + 1) * sizeof(int32_t);
      int32_t* cu_seqlens_d = reinterpret_cast<int32_t*>(tmp->mut_dptr());
      half* packed_qkv =
          reinterpret_cast<half*>(tmp->mut_dptr<char>() + GetCudaAlignedSize(cu_seqlens_d_size));
      constexpr int pack_size = 4;
      using PackType = Pack<half, pack_size>;
      int count = batch_size * query_seq_len * query_hidden_size * 3 / pack_size;
      PackQkv<PackType><<<(count - 1 + 256) / 256, 256, 0, cuda_stream->cuda_stream()>>>(
          batch_size, query_seq_len, num_heads, query_head_size / pack_size,
          reinterpret_cast<const PackType*>(query->dptr()),
          reinterpret_cast<const PackType*>(key->dptr()),
          reinterpret_cast<const PackType*>(value->dptr()), reinterpret_cast<PackType*>(packed_qkv),
          cu_seqlens_d);

      nvinfer1::plugin::FusedMultiHeadFlashAttentionKernel const* kernels =
          nvinfer1::plugin::getFMHACubinKernels(nvinfer1::plugin::DATA_TYPE_FP16, arch);
      run_fmha_v2_api(packed_qkv, cu_seqlens_d, out->mut_dptr(), batch_size * query_seq_len, arch,
                      kernels, batch_size, num_heads, query_head_size, query_seq_len,
                      cuda_stream->cuda_stream());
      return;
    }

    Params params{};
    params.data_type = data_type;
    params.num_batches = batch_size;
    params.num_heads = num_heads;
    params.query_seq_len = query_seq_len;
    params.kv_seq_len = kv_seq_len;
    params.head_size = query_hidden_size / num_heads;
    params.value_head_size = value_hidden_size / num_heads;
    params.query_hidden_stride = query->shape_view().At(2);
    params.key_hidden_stride = key->shape_view().At(2);
    params.value_hidden_stride = value->shape_view().At(2);
    params.query_ptr = query->dptr<char>() + query_hidden_offset;
    params.key_ptr = key->dptr<char>() + key_hidden_offset;
    params.value_ptr = value->dptr<char>() + value_hidden_offset;
    params.out_ptr = out->mut_dptr();
    const int64_t tmp_buffer_size = tmp->shape_view().elem_cnt();
    params.workspace = tmp->mut_dptr<char>();
    params.workspace_size = tmp_buffer_size;
    params.causal = causal;
    DispatchCutlassFmha(params, cuda_stream);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

size_t InferTmpBufferSize(InferContext* ctx) {
  const auto& out_desc = ctx->OutputTensorDesc("out", 0);
  size_t buffer_size = 0;
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(DataType::kFloat));
  buffer_size +=
      GetCudaAlignedSize(out_desc.shape().elem_cnt() * GetSizeOfDataType(out_desc.data_type())) * 3;
  buffer_size +=
      GetCudaAlignedSize((out_desc.shape().At(0) + 1) * GetSizeOfDataType(DataType::kInt32));
  return buffer_size;
}

}  // namespace

#define REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(dtype)    \
  REGISTER_USER_KERNEL("fused_multi_head_attention_inference")         \
      .SetCreateFn<FusedMultiHeadAttentionInferenceKernel>()           \
      .SetIsMatchedHob((user_op::HobDeviceType() == DeviceType::kCUDA) \
                       && (user_op::HobDataType("out", 0) == dtype))   \
      .SetInferTmpSizeFn(InferTmpBufferSize);

REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat16)
REGISTER_FUSED_MULTI_HEAD_ATTENTION_INFERENCE_KERNEL(DataType::kFloat)

}  // namespace user_op

}  // namespace oneflow

#endif  // WITH_CUTLASS
